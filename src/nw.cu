#include "hip/hip_runtime.h"
#ifndef NW_CUH
#include <iostream>
#include <stdio.h>
#include "nw.cuh"
#endif

extern int xdropval;
void printGpuProperties () {
    int nDevices;

    // Store the number of available GPU device in nDevicess
    hipError_t err = hipGetDeviceCount(&nDevices);

    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: hipGetDeviceCount failed!\n");
        exit(1);
    }

    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Device memory: %lu\n", prop.totalGlobalMem);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }
}


void NWGPU::DeviceArrays::allocateDeviceArrays(
    size_t h_numAlignments,
    std::vector<char *>& h_refSeq,
    std::vector<char *>& h_querySeq,
    std::vector<size_t>& h_refLen,
    std::vector<size_t>& h_queryLen,
    Params& param
){

    hipError_t err;

    d_numAlignments = h_numAlignments;
    d_match = param.match;
    d_mismatch = param.mismatch;
    d_gapOpen = param.gapOpen;

    
    size_t * refLen = new size_t[h_numAlignments];
    size_t * queryLen = new size_t[h_numAlignments];

    size_t * refStartCord = new size_t[h_numAlignments];
    size_t * queryStartCord = new size_t[h_numAlignments];


    size_t totalRefLength = 0;
    size_t totalQueryLength = 0;

    // Allocate Ref and Query Length memory
    err = hipMalloc(&d_refLen, h_numAlignments*sizeof(size_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_queryLen, h_numAlignments*sizeof(size_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    // Transfer Sequence Length values

    refStartCord[0]=0; queryStartCord[0]=0;
    for (size_t i=0; i<h_numAlignments; i++)
    {
        refLen[i] = h_refLen[i];
        queryLen[i] = h_queryLen[i];
        totalRefLength += h_refLen[i];
        totalQueryLength += h_queryLen[i];
        if (i < h_numAlignments - 1)
        {
            refStartCord[i+1] = totalRefLength;
            queryStartCord[i+1] = totalQueryLength;
        }

    }


    err = hipMemcpy(d_refLen, refLen, h_numAlignments*sizeof(size_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_queryLen, queryLen, h_numAlignments*sizeof(size_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    // Allocate Start Coordinate Memory
    err = hipMalloc(&d_refStartCord, h_numAlignments*sizeof(size_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_queryStartCord, h_numAlignments*sizeof(size_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    // Transfer Start Coordinate  Data
    err = hipMemcpy(d_refStartCord, refStartCord, h_numAlignments*sizeof(size_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_queryStartCord, queryStartCord, h_numAlignments*sizeof(size_t), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }


    // Allocate Ref and Query Sequence memory
    err = hipMalloc(&d_refSeq, totalRefLength*sizeof(char));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    err = hipMalloc(&d_querySeq, totalQueryLength*sizeof(char));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    // Transfer Sequences
    char * ref = new char[totalRefLength];
    char * query = new char[totalQueryLength];

    uint32_t refIndex=0, queryIndex=0;

    for (size_t i=0; i<h_numAlignments; i++)
    {
        char * currentRef =  h_refSeq[i];
        char * currentQuery =  h_querySeq[i];
        size_t currentRefLength = h_refLen[i];
        size_t currentQueryLength = h_queryLen[i];
        for (size_t j=0; j<currentRefLength; j++)
            ref[refIndex++] = currentRef[j];
        for (size_t j=0; j<currentQueryLength; j++)
            query[queryIndex++] = currentQuery[j];
    }

    err = hipMemcpy(d_refSeq, ref, totalRefLength*sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    err = hipMemcpy(d_querySeq, query, totalQueryLength*sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }


    // Allocate memory for the output
    err = hipMalloc(&d_scores, h_numAlignments*sizeof(uint32_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    // Allocate memory to store traceback pointers
    err = hipMalloc(&d_tbPointers, h_numAlignments*(512)*sizeof(int8_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    // Allocate memory to store traceback Length
    err = hipMalloc(&d_tbPointersLen, h_numAlignments*sizeof(uint32_t));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Gpu_ERROR: hipMalloc failed!\n");
        exit(1);
    }

    hipDeviceSynchronize();

}

void NWGPU::DeviceArrays::deallocateDeviceArrays()
{
    hipFree(d_refSeq);
    hipFree(d_querySeq);
    hipFree(d_refLen);
    hipFree(d_queryLen);
    hipFree(d_refStartCord);
    hipFree(d_queryStartCord);
    hipFree(d_tbPointers);
    hipFree(d_tbPointersLen);
}


__device__ void tracebackSeqtoSeq
(
    int8_t tbMatrix[],
    int32_t tbIdx,
    int32_t wfLL[],
    int32_t wfLen[],
    size_t refLen,
    size_t queryLen,
    int8_t * tbPointers,
    int32_t * tbPointersLen
){
    int32_t refIndex = refLen-1;
    int32_t queryIndex = queryLen-1;

    tbIdx--;
    
    int32_t k=refLen+queryLen;

    int8_t state;
    int32_t currentTbPointersIdx = 0;


    while (k>=0)
    {
        state = tbMatrix[tbIdx] & 0x03;
        tbIdx -= (refIndex - wfLL[k] + 1 + wfLen[k-1]);

        tbPointers[currentTbPointersIdx++] = state;
        if (state == 0) 
        {
            tbIdx -= wfLen[k-2]; tbIdx += refIndex - wfLL[k-2]; k--;k--;
        }
        else if (state == 1) 
        {
            tbIdx += (refIndex - wfLL[k-1] + 1); k--;
        }
        else
        {
            tbIdx += (refIndex - wfLL[k-1]); k--; 
        }
    }

    tbPointersLen[0] = currentTbPointersIdx;

}
/*
            int ref_index = currentRefStartCord + tx;
            int query_index = currentQueryStartCord + tx;         

            //char * ref = d_ref + currentRefStartCord;
            //char * query = d_query + currentQueryStartCord;
            //    Create shared memory for query and reference
            __shared__ char ref[256];
            __shared__ char query[256];

                        if(tx<currentRefLength){
                ref[tx] = d_ref[ref_index];
            }
                    __syncthreads();
                    if(tx<currentQueryLength){
                        query[tx] = d_query[query_index];
                    }
                    __syncthreads();
                    */

__device__ int32_t max3(int32_t a, int32_t b, int32_t c) {
    return max(max(a, b), c);
}

__inline__ __device__ void warpReduce(volatile int32_t *input,
										  int myTId){
		input[myTId] = (input[myTId] > input[myTId + 32]) ? input[myTId] : input[myTId + 32]; 
		input[myTId] = (input[myTId] > input[myTId + 16]) ? input[myTId] : input[myTId + 16];
		input[myTId] = (input[myTId] > input[myTId + 8]) ? input[myTId] : input[myTId + 8]; 
		input[myTId] = (input[myTId] > input[myTId + 4]) ? input[myTId] : input[myTId + 4];
		input[myTId] = (input[myTId] > input[myTId + 2]) ? input[myTId] : input[myTId + 2];
		input[myTId] = (input[myTId] > input[myTId + 1]) ? input[myTId] : input[myTId + 1];
}

__inline__ __device__ int32_t reduce_max(int32_t *input, int32_t dim, int n_threads){
	unsigned int myTId = threadIdx.x;   
	if(dim>32){
		for(int i = n_threads/2; i >32; i>>=1){
			if(myTId < i){
						input[myTId] = (input[myTId] > input[myTId + i]) ? input[myTId] : input[myTId + i];
			}__syncthreads();
		}
	}
	if(myTId<32)
		warpReduce(input, myTId);
	__syncthreads();
	return input[0];
}


//With Xdrop Parallel
__global__ void alignSeqToSeq
(
    size_t d_numAlignments,
    char* d_ref,
    char* d_query,
    size_t * refLen,
    size_t * queryLen,
    size_t * refStartCord,
    size_t * queryStartCord,
    int matchPoints, 
    int mismatchPoints,
    int gapOpenPoints,
    int * d_scores,
    int8_t * tbPointers,
    int32_t * tbPointersLen,
    int *Xdrop_value
){

    int tx = threadIdx.x;
    int bx = blockIdx.x;

    int bs = blockDim.x;
    int gs = gridDim.x;
    int tid = threadIdx.x + blockIdx.x*blockDim.x;

        for (size_t n= blockIdx.x; n<d_numAlignments; n+= gridDim.x)
        {
            size_t currentRefLength = refLen[n];
            size_t currentQueryLength = queryLen[n];

            size_t currentRefStartCord = refStartCord[n];
            size_t currentQueryStartCord = queryStartCord[n];

            int ref_index = currentRefStartCord + tx;
            int query_index = currentQueryStartCord + tx;   
            __shared__ int32_t max_seen_antidiag;
            __shared__ int32_t max_seen_current;
                if(tx==0){
                    max_seen_antidiag = -INF;    
                    max_seen_current = -INF;  
                }
            
            //    Create shared memory for query and reference, so that we don't have to worry about the memory coalsced
            __shared__ char ref[256];
            __shared__ char query[256];

                        if(tx<currentRefLength){
                ref[tx] = d_ref[ref_index];
            }
                    if(tx<currentQueryLength){
                        query[tx] = d_query[query_index];
                    }
            
            
            int8_t * currentTbPointers = tbPointers + n*512;
            int32_t * currentTbPointersLen = tbPointersLen + n;

            size_t maxWFLen = currentRefLength + currentQueryLength + 2; //wavefront length

           __shared__ int32_t score[500];
           int32_t offset = 0;
           int32_t k = 0;
            for (size_t i=tx;i<500;i+=bs) {score[i] = 0;}
            int32_t maxScore = 0;

            __shared__ int32_t H[3][500];
            __shared__ int32_t temp_h3[500];
           
            int32_t L[3], U[3];

            int32_t wfLL[256*2+2];
            int32_t wfLen[256*2+2];
            int8_t tbMatrix[258*258]; //(256+2)^2
            int32_t tbIdx = 0;

            int8_t state=0;

            for(size_t i=0; i<3; i++)
            {
                L[i]=0; U[i]=0;
            }
           for (size_t i=0; i<3; i++)
            {
                for (size_t j=tx; j<500; j+=bs) {H[i][j] = 0;
                temp_h3[j] = 0;
                }
            }
            __syncthreads();    
            for (int32_t k=0; k<currentRefLength+currentQueryLength+1; k++)
            {
                L[k%3] = (k<=currentQueryLength)?0:k-currentQueryLength;
                U[k%3] = (k<=currentRefLength)?k:currentRefLength;
                wfLL[k] = L[k%3];
                wfLen[k] = U[k%3]-L[k%3]+1;
                if(L[k%3]+tx<U[k%3]+1)
                {   
                    int32_t i = L[k%3] + tx;
                    int32_t j=(k-i); //j->Query Index
                    int32_t match = -INF, insOp = -INF, delOp = -INF;
                    offset = i-L[k%3];
                    int32_t offsetDiag = L[k%3]-L[(k+1)%3]+offset-1;
                    int32_t offsetUp = L[k%3]-L[(k+2)%3]+offset;
                    int32_t offsetLeft = L[k%3]-L[(k+2)%3]+offset-1;
                   
                    if (k==0) match = 0;
                    
                    if (offsetDiag>=0 && i-1>=0 && j-1>=0)
                    {
                        char refVal = ref[i-1];
                        char queryVal = query[j-1];
                        if (refVal == queryVal) match = H[(k+1)%3][offsetDiag] + matchPoints;
                        else match = H[(k+1)%3][offsetDiag] + mismatchPoints;
                    }
                    if (offsetUp >= 0)
                        insOp = H[(k+2)%3][offsetUp] + gapOpenPoints;
                    if (offsetLeft >=0)
                        delOp = H[(k+2)%3][offsetLeft] + gapOpenPoints;
                     
                    H[k%3][offset] = max3(insOp,delOp,match);
                   temp_h3[offset] = H[k%3][offset];
                    score[offset] = H[k%3][offset];
                }
                __syncthreads();    //Wait for all threads to update the H array
                //Use reduction to find max in current diagnol
            for(int stride = bs / 2; stride > 0; stride >>= 1) {
                    if (tx < stride) {
                        int idx1 = tx;
                        int idx2 = tx + stride;
                        temp_h3[idx1] = max(temp_h3[idx1], temp_h3[idx2]);
                    }
                    // Wait for all threads to complete their operation at this stride
                    __syncthreads();
                 }
                   if(tx==0){
                       max_seen_current = temp_h3[0];
                   }
                if(tx==0){
                if(max_seen_current<=max_seen_antidiag-(*Xdrop_value))
                {
                    score[offset] = max_seen_antidiag;
                    if(bx==4)
                    printf("Breaking at Diagnoal %d \n\n",k);
                    break;
                    
                }
                else
                {
                    max_seen_antidiag = (max_seen_current>max_seen_antidiag)?max_seen_current:max_seen_antidiag;
                }
                }               
            }
            if(tx==0)       //Thread 0 of each block updates its scores to d_scores
            d_scores[n] = score[offset];
        }
    }



/*
//With Xdrop serial implementation
__global__ void alignSeqToSeq
(
    size_t d_numAlignments,
    char* d_ref,
    char* d_query,
    size_t * refLen,
    size_t * queryLen,
    size_t * refStartCord,
    size_t * queryStartCord,
    int matchPoints, 
    int mismatchPoints,
    int gapOpenPoints,
    int * d_scores,
    int8_t * tbPointers,
    int32_t * tbPointersLen,
    int *Xdrop_value
){

    int tx = threadIdx.x;
    int bx = blockIdx.x;

    int bs = blockDim.x;
    int gs = gridDim.x;
    int tid = threadIdx.x + blockIdx.x*blockDim.x;

	if(tx==0 && bx==0){


        for (size_t n= 0; n<d_numAlignments; n++)
        {
            int32_t max_seen_antidiag = -INF;    
            int32_t max_seen_current = -INF; 

           size_t currentRefLength = refLen[n];
            size_t currentQueryLength = queryLen[n];

            size_t currentRefStartCord = refStartCord[n];
            size_t currentQueryStartCord = queryStartCord[n];

            char * ref = d_ref + currentRefStartCord;
            char * query = d_query + currentQueryStartCord;
            
            int8_t * currentTbPointers = tbPointers + n*512;
            int32_t * currentTbPointersLen = tbPointersLen + n;

            size_t maxWFLen = currentRefLength + currentQueryLength + 2; //wavefront length

            __shared__ int32_t score[500];
            for (size_t i = 0;i<500;i++) score[i] = 0;
            int32_t maxScore = 0;

            int32_t H[3][500];
            int32_t L[3], U[3];

            int32_t wfLL[256*2+2];
            int32_t wfLen[256*2+2];
            int8_t tbMatrix[258*258]; //(256+2)^2
            int32_t tbIdx = 0;
               int8_t state=0;
            for(size_t i=0;i<(258*258);i++){
                    tbMatrix[i] = 2;
            }
         

            for(size_t i=0; i<3; i++)
            {
                L[i]=0; U[i]=0;
            }

            for (size_t i=0; i<3; i++)
            {
                for (size_t j=0; j<500; j++) H[i][j] = 0;
            }

   
                int32_t offset = 0;
            for (int32_t k=0; k<currentRefLength+currentQueryLength+1; k++)
            {
                L[k%3] = (k<=currentQueryLength)?0:k-currentQueryLength;
                U[k%3] = (k<=currentRefLength)?k:currentRefLength;
                wfLL[k] = L[k%3];
                wfLen[k] = U[k%3]-L[k%3]+1;
                int32_t max_temp = -INF;
                for(int32_t i=L[k%3]; i<U[k%3]+1; i++) // i -> Reference Index
                {   
                    int32_t j=(k-i); //j->Query Index
                    int32_t match = -INF, insOp = -INF, delOp = -INF;
                    offset = i-L[k%3];
                    int32_t offsetDiag = L[k%3]-L[(k+1)%3]+offset-1;
                    int32_t offsetUp = L[k%3]-L[(k+2)%3]+offset;
                    int32_t offsetLeft = L[k%3]-L[(k+2)%3]+offset-1;


                    if (k==0) match = 0;
                    
                    if (offsetDiag>=0)
                    {
                        char refVal = ref[i-1];
                        char queryVal = query[j-1];
                        if (refVal == queryVal) match = H[(k+1)%3][offsetDiag] + matchPoints;
                        else match = H[(k+1)%3][offsetDiag] + mismatchPoints;
                    }
                    
                    if (offsetUp >= 0)
                        insOp = H[(k+2)%3][offsetUp] + gapOpenPoints;

                    if (offsetLeft >=0)
                        delOp = H[(k+2)%3][offsetLeft] + gapOpenPoints;

                    
                        H[k%3][offset] = max3(match,insOp,delOp);
                        score[offset] = H[k%3][offset];
                    
                }
                 for(int32_t i=L[k%3]; i<U[k%3]+1; i++) // i -> Reference Index
                {
                 int td = i-L[k%3];
                              if(H[k%3][td]>max_temp)
                            max_temp = H[k%3][td];
                    }
                    max_seen_current = max_temp;
                if(max_seen_current<=max_seen_antidiag-(*Xdrop_value))
                {
                    score[offset] = max_seen_antidiag;
                    break;
                    
                }
                else
                {
                    max_seen_antidiag = (max_seen_current>max_seen_antidiag)?max_seen_current:max_seen_antidiag;
                }
                
            }
            d_scores[n] = score[offset];
        }
    }
}
*/

/*
// Without Xdrop
__global__ void alignSeqToSeq
(
    size_t d_numAlignments,
    char* d_ref,
    char* d_query,
    size_t * refLen,
    size_t * queryLen,
    size_t * refStartCord,
    size_t * queryStartCord,
    int matchPoints, 
    int mismatchPoints,
    int gapOpenPoints,
    int * d_scores,
    int8_t * tbPointers,
    int32_t * tbPointersLen
){

    int tx = threadIdx.x;
    int bx = blockIdx.x;

    int bs = blockDim.x;
    int gs = gridDim.x;
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
        for (size_t n= blockIdx.x; n<d_numAlignments; n+= gridDim.x)
        {
            size_t currentRefLength = refLen[n];
            size_t currentQueryLength = queryLen[n];

            size_t currentRefStartCord = refStartCord[n];
            size_t currentQueryStartCord = queryStartCord[n];

            int ref_index = currentRefStartCord + tx;
            int query_index = currentQueryStartCord + tx;         

            //    Create shared memory for query and reference
            __shared__ char ref[256];
            __shared__ char query[256];

                        if(tx<currentRefLength){
                ref[tx] = d_ref[ref_index];
            }
                    __syncthreads();
                    if(tx<currentQueryLength){
                        query[tx] = d_query[query_index];
                    }
                    __syncthreads();
            
            
            int8_t * currentTbPointers = tbPointers + n*512;
            int32_t * currentTbPointersLen = tbPointersLen + n;

            size_t maxWFLen = currentRefLength + currentQueryLength + 2; //wavefront length

            __shared__ int32_t score[500];
            for (size_t i=tx;i<500;i+=bs) {score[i] = 0; }//insOp[i] = -INF; delOp[i] = -INF;}
            __syncthreads();
            int32_t maxScore = 0;

            __shared__ int32_t H[3][500];
           
            int32_t L[3], U[3];

            int32_t wfLL[256*2+2];
            int32_t wfLen[256*2+2];
            int8_t tbMatrix[258*258]; //(256+2)^2
            int32_t tbIdx = 0;

            int8_t state=0;

            for(size_t i=tx; i<3; i+=bs)
            {
                L[i]=0; U[i]=0;
            }
                __syncthreads();
           for (size_t i=0; i<3; i++)
            {
                for (size_t j=tx; j<500; j+=bs) H[i][j] = 0;
            }

            __syncthreads();
                int32_t offset = 0;
            for (int32_t k=0; k<currentRefLength+currentQueryLength+1; k++)
            {
                L[k%3] = (k<=currentQueryLength)?0:k-currentQueryLength;
                U[k%3] = (k<=currentRefLength)?k:currentRefLength;
                wfLL[k] = L[k%3];
                wfLen[k] = U[k%3]-L[k%3]+1;

              //for(int32_t i=L[k%3]+tx; i<U[k%3]+1; i+=bs) // i -> Reference Index
              if(L[k%3]+tx<U[k%3]+1)
                {   
                    int32_t i = L[k%3]+tx;
                    int32_t j=(k-i); //j->Query Index
                    int32_t match = -INF, insOp = -INF, delOp = -INF;
                    offset = i-L[k%3];
                    int32_t offsetDiag = L[k%3]-L[(k+1)%3]+offset-1;
                    int32_t offsetUp = L[k%3]-L[(k+2)%3]+offset;
                    int32_t offsetLeft = L[k%3]-L[(k+2)%3]+offset-1;
                   
                    if (k==0) match = 0;
                    
                    if (offsetDiag>=0 && i-1>=0 && j-1>=0)
                    {
                        char refVal = ref[i-1];
                        char queryVal = query[j-1];
                        if (refVal == queryVal) match = H[(k+1)%3][offsetDiag] + matchPoints;
                        else match = H[(k+1)%3][offsetDiag] + mismatchPoints;
                    }
                    //__syncthreads();
                    if (offsetUp >= 0)
                        insOp = H[(k+2)%3][offsetUp] + gapOpenPoints;
                    //__syncthreads();
                    if (offsetLeft >=0)
                        delOp = H[(k+2)%3][offsetLeft] + gapOpenPoints;
                    //__syncthreads();

                     
                    H[k%3][offset] = max3(insOp,delOp,match);
                   //__syncthreads();
                    score[offset] = H[k%3][offset];
                   // __syncthreads();
                }
                __syncthreads();
            }
            if(tx==0)       //Thread 0 of each block updates its scores to d_scores
            d_scores[n] = score[offset];
        }
    }
*/



void NWGPU::NWonGPU
(
    size_t d_numAlignments,
    char * d_ref,
    char * d_query,
    size_t * d_refLen,
    size_t * d_queryLen,
    size_t * d_refStartCord,
    size_t * d_queryStartCord,
    int d_match,
    int d_mismatch,
    int d_gapOpen,
    int * d_scores,
    int8_t * d_tbPointers,
    int32_t * d_tbPointersLen
){

    int blockPerGrid = 1024;
    int threadsPerBlock = 256;
    int *d_xdrop;
    int xdrop_value = xdropval;
  //  printf("Thge xdrop value = %d\n",xdropval);
    hipMalloc(&d_xdrop,sizeof(int));
    hipMemcpy(d_xdrop,&xdrop_value,sizeof(int),hipMemcpyHostToDevice);

   alignSeqToSeq<<<blockPerGrid,threadsPerBlock>>>( d_numAlignments,
                            d_ref,
                            d_query,
                            d_refLen,
                            d_queryLen,
                            d_refStartCord,
                            d_queryStartCord,
                            d_match,
                            d_mismatch,
                            d_gapOpen,
                            d_scores,
                            d_tbPointers,
                            d_tbPointersLen,
			    d_xdrop);

        /*        

             alignSeqToSeq<<<blockPerGrid,threadsPerBlock>>>( d_numAlignments,
                            d_ref,
                            d_query,
                            d_refLen,
                            d_queryLen,
                            d_refStartCord,
                            d_queryStartCord,
                            d_match,
                            d_mismatch,
                            d_gapOpen,
                            d_scores,
                            d_tbPointers,
                            d_tbPointersLen);
                
*/
    hipDeviceSynchronize();

}

void NWGPU::DeviceArrays::printScores(size_t h_numAlignments)
{
    hipError_t err;
    int* h_scores = new int[h_numAlignments];

    err = hipMemcpy(h_scores, d_scores, h_numAlignments*sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    printf("Count\tScore\n");
    for (size_t i=0; i<h_numAlignments; i++)
    {
        printf("%zu\t%d\n",i,h_scores[i]);
    }
}

void NWGPU::DeviceArrays::printTbPointers(size_t h_numAlignments)
{
    hipError_t err;
    int8_t* tbPointers = new int8_t[h_numAlignments*512];
    int32_t* tbPointersLen = new int32_t[h_numAlignments];

   // err = hipMemcpy(tbPointers, d_tbPointers, h_numAlignments*512*sizeof(int8_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

   //err = hipMemcpy(tbPointersLen, d_tbPointersLen, h_numAlignments*sizeof(int32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "GPU_ERROR: cudaMemCpy failed!\n");
        exit(1);
    }

    for (size_t i=0; i<h_numAlignments; i++)
    {
        int8_t * currentTbPointers = tbPointers + 512*i;
    //    printf ("%d\t", i);
        for (size_t j=0; j<tbPointersLen[i]; j++)
        {
      //      printf("%d ", currentTbPointers[j]);
        }
       // printf("\n");
    }
}


